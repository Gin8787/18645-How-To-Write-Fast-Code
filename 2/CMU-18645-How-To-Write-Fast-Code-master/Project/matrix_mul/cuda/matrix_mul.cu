#include "hip/hip_runtime.h"
/*
    Copyright (C) 2011  Abhinav Jauhri (abhinav.jauhri@gmail.com), Carnegie Mellon University - Silicon Valley 

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix_mul.h"
#include <math.h>
#define TILE_WIDTH 32


namespace cuda
{
  __global__ 
  void 
  matrix_mul_kernel1(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, int sq_dimension)
  {
    
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;

    float sum = 0;
    
	if(col < sq_dimension && row < sq_dimension) {
    	for(int k = 0; k < sq_dimension; k++)
      	{
			sum += sq_matrix_1[row*sq_dimension + k] * sq_matrix_2[k*sq_dimension + col];
      	}
    	sq_matrix_result[row*sq_dimension + col] = sum;
	}
  
  }

  __global__
  void
  matrix_mul_kernel2(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, int sq_dimension)
  {

    __shared__ float matA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float matB[TILE_WIDTH][TILE_WIDTH];
    
	int bx = blockIdx.x;
	int by = blockIdx.y;
    int tx = threadIdx.x;
	int ty = threadIdx.y;

    int col = bx*TILE_WIDTH + tx;
    int row = by*TILE_WIDTH + ty;
   
    float sum = 0;    
	int index1, index2;

    for(int m = 0; m < gridDim.x; ++m) {
		
		index1 = m*TILE_WIDTH+tx;
		index2 = m*TILE_WIDTH+ty;		
		if(row < sq_dimension && index1 < sq_dimension) {
        	matA[ty][tx] = sq_matrix_1[row*sq_dimension + index1];
		}
		else {
			matA[ty][tx] = 0;
		}
		if(col < sq_dimension && index2 < sq_dimension) {
        	matB[ty][tx] = sq_matrix_2[index2*sq_dimension + col];
		}
		else {
			matB[ty][tx] = 0;
		}
		__syncthreads();

//		#pragma unroll
        for(int k = 0; k < TILE_WIDTH; ++k) {
            sum += matA[ty][k]*matB[k][tx];  
        }
        __syncthreads();
    }
    
	if(row < sq_dimension && col < sq_dimension)
    	sq_matrix_result[row*sq_dimension + col] = sum;

  }

  __global__
  void
  matrix_mul_kernel3(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, int sq_dimension) //loop unrolling
  { 
    __shared__ float matA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float matB[TILE_WIDTH][TILE_WIDTH];
    
	int bx = blockIdx.x;
	int by = blockIdx.y;
    int tx = threadIdx.x;
	int ty = threadIdx.y;

    int col = bx*TILE_WIDTH + tx;
    int row = by*TILE_WIDTH + ty;
   
    float sum = 0;    
	int index1, index2;

    for(int m = 0; m < gridDim.x; ++m) {
		
		index1 = m*TILE_WIDTH+tx;
		index2 = m*TILE_WIDTH+ty;		
		if(row < sq_dimension && index1 < sq_dimension) {
        	matA[ty][tx] = sq_matrix_1[row*sq_dimension + index1];
		}
		else {
			matA[ty][tx] = 0;
		}
		if(col < sq_dimension && index2 < sq_dimension) {
        	matB[ty][tx] = sq_matrix_2[index2*sq_dimension + col];
		}
		else {
			matB[ty][tx] = 0;
		}
		__syncthreads();


        sum += matA[ty][0]*matB[0][tx] + matA[ty][1]*matB[1][tx] + matA[ty][2]*matB[2][tx] + matA[ty][3]*matB[3][tx] + matA[ty][4]*matB[4][tx] +
		      matA[ty][5]*matB[5][tx] + matA[ty][6]*matB[6][tx] + matA[ty][7]*matB[7][tx] + matA[ty][8]*matB[8][tx] + matA[ty][9]*matB[9][tx] +
			  matA[ty][10]*matB[10][tx] + matA[ty][11]*matB[11][tx] + matA[ty][12]*matB[12][tx] + matA[ty][13]*matB[13][tx] + matA[ty][14]*matB[14][tx] +
			  matA[ty][15]*matB[15][tx] + matA[ty][16]*matB[16][tx] + matA[ty][17]*matB[17][tx] + matA[ty][18]*matB[18][tx] + matA[ty][19]*matB[19][tx] +
			  matA[ty][20]*matB[20][tx] + matA[ty][21]*matB[21][tx] + matA[ty][22]*matB[22][tx] + matA[ty][23]*matB[23][tx] + matA[ty][24]*matB[24][tx] +
			  matA[ty][25]*matB[25][tx] + matA[ty][26]*matB[26][tx] + matA[ty][27]*matB[27][tx] + matA[ty][28]*matB[28][tx] + matA[ty][29]*matB[29][tx] +
			  matA[ty][30]*matB[30][tx] + matA[ty][31]*matB[31][tx];

        __syncthreads();
    }
    
	if(row < sq_dimension && col < sq_dimension)
    	sq_matrix_result[row*sq_dimension + col] = sum;
  
  }
  
   __global__
  void
  matrix_mul_kernel4(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, int sq_dimension) // use traverse and avoid shared memory bank conflict
  {

    __shared__ float matA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float matB[TILE_WIDTH][TILE_WIDTH];  //add one column to avoid bank conflict
    
	int bx = blockIdx.x;
	int by = blockIdx.y;
    int tx = threadIdx.x;
	int ty = threadIdx.y;

    int col = bx*TILE_WIDTH + tx;
    int row = by*TILE_WIDTH + ty;
   
    float sum = 0;    
	int index1, index2;

    for(int m = 0; m < gridDim.x; ++m) {
		
		index1 = m*TILE_WIDTH+tx;
		index2 = m*TILE_WIDTH+ty;		
		if(row < sq_dimension && index1 < sq_dimension) {
        	matA[ty][tx] = sq_matrix_1[row*sq_dimension + index1];
		}
		else {
			matA[ty][tx] = 0;
		}
		if(col < sq_dimension && index2 < sq_dimension) {
        	matB[tx][ty] = sq_matrix_2[index2*sq_dimension + col];
		}
		else {
			matB[tx][ty] = 0;
		}
		__syncthreads();

		//#pragma unroll
        for(int k = 0; k < TILE_WIDTH; ++k) {
            sum += matA[ty][k]*matB[tx][k];  
        }
        __syncthreads();
    }
    
	if(row < sq_dimension && col < sq_dimension)
    	sq_matrix_result[row*sq_dimension + col] = sum;

  }


  void 
  matrix_multiplication(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, unsigned int sq_dimension)
  {
    
	int choice = 1;   // choose the method

	int size = sq_dimension * sq_dimension * sizeof(float);
    float *sq_matrix_1_d, *sq_matrix_2_d, *sq_matrix_result_d;
    
    /***************************************************
  1st Part: Allocation of memory on device memory  
    ****************************************************/
    
    /* copy sq_matrix_1 and sq_matrix_2 to device memory */
    hipMalloc((void**) &sq_matrix_1_d, size);
    hipMemcpy(sq_matrix_1_d, sq_matrix_1, size, hipMemcpyHostToDevice);
    hipMalloc((void**) &sq_matrix_2_d, size);
    hipMemcpy(sq_matrix_2_d, sq_matrix_2, size, hipMemcpyHostToDevice);
    
    /*allocate sq_matrix_result on host */
    hipMalloc((void**) &sq_matrix_result_d, size);
    
    /***************************************************
   2nd Part: Inovke kernel 
    ****************************************************/
	if(choice == 0)  {    //use the block with global memory  
		if(sq_dimension <= TILE_WIDTH) {
			dim3 dimBlock(sq_dimension, sq_dimension);
			dim3 dimGrid(1,1);
			matrix_mul_kernel1<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d,sq_dimension);		
		}
		else {
			dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
			int blockNum = ceil(sq_dimension*1.0/TILE_WIDTH);
			dim3 dimGrid(blockNum, blockNum);
			matrix_mul_kernel1<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
		}
  
	}

	else if(choice == 1) {  //use the block with shared memory
		if(sq_dimension <= TILE_WIDTH) {
        	dim3 dimBlock(sq_dimension, sq_dimension);
			dim3 dimGrid(1,1);
    		matrix_mul_kernel2<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);    		
    	}
    	else {
			dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
			int blockNum = ceil(sq_dimension*1.0/TILE_WIDTH);
    		dim3 dimGrid(blockNum,blockNum);
    		matrix_mul_kernel2<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
    	}
	}
	
	else if(choice == 2) {  //use loop unrolling
		if(sq_dimension <= TILE_WIDTH) {
        	dim3 dimBlock(sq_dimension, sq_dimension);
			dim3 dimGrid(1,1);
    		matrix_mul_kernel3<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);    		
    	}
    	else {
			dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
			int blockNum = ceil(sq_dimension*1.0/TILE_WIDTH);
    		dim3 dimGrid(blockNum,blockNum);
    		matrix_mul_kernel3<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
    	}
	}
	
	else if(choice == 3) {  //use traverse
		if(sq_dimension <= TILE_WIDTH) {
        	dim3 dimBlock(sq_dimension, sq_dimension);
			dim3 dimGrid(1,1);
    		matrix_mul_kernel4<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);    		
    	}
    	else {
			dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
			int blockNum = ceil(sq_dimension*1.0/TILE_WIDTH);
    		dim3 dimGrid(blockNum,blockNum);
    		matrix_mul_kernel4<<<dimGrid, dimBlock>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
    	}
	}
    
    /***************************************************
   3rd Part: Transfer result from device to host 
    ****************************************************/
    hipMemcpy(sq_matrix_result, sq_matrix_result_d, size, hipMemcpyDeviceToHost);
    hipFree(sq_matrix_1_d);
    hipFree(sq_matrix_2_d);
    hipFree(sq_matrix_result_d);
  }  
} // namespace cuda
